
#include <hip/hip_runtime.h>
__global__ 
void sparseMatrixVectorSetFlags(
                                unsigned int             *d_flags, 
                                const unsigned int       *d_rowindx, 
                                unsigned int             numRows
                                )
{
    unsigned int iGlobal = (blockIdx.x * (blockDim.x << 3)) + threadIdx.x;

    bool isLastBlock = (blockIdx.x == (gridDim.x-1));

    for (unsigned int i = 0; i < 8; ++i)
    {
        if (isLastBlock)
        {
            if (iGlobal < numRows)
            {
                d_flags[d_rowindx[iGlobal]] = 1;
            }
        }
        else
        {
            d_flags[d_rowindx[iGlobal]] = 1;
        }

        iGlobal += blockDim.x;
    }
    
    __syncthreads();
}
