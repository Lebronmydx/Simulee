
#include <hip/hip_runtime.h>
#define NUM_RND_BLOCKS                      96
#define NUM_RND_THREADS_PER_BLOCK           128
#define NUM_RND_STREAMS                     (NUM_RND_BLOCKS * NUM_RND_THREADS_PER_BLOCK)

/*
 * Defines for getting the values at the lower and upper 32 bits
 * of a 64-bit number.
 */
#define LOW_BITS(x)                         ((x) & 0xffffffff)
#define HIGH_BITS(x)                        ((x) >> 32)

/*
 * Number of iterations to run random number generator upon initialization.
 */
#define NUM_RND_BURNIN                      100

/*
 * CUDA grid dimensions for different types of kernels
 */
#define COPY_BLOCK_SIZE                     16
#
// element-wise kernels use min(ceil(N / 512), 4096) blocks of 512 threads
#define MAX_VECTOR_OP_BLOCKS                4096
#define MAX_VECTOR_OP_THREADS_PER_BLOCK     512
#define NUM_VECTOR_OP_BLOCKS(N)             (min(((N) + MAX_VECTOR_OP_THREADS_PER_BLOCK - 1)/MAX_VECTOR_OP_THREADS_PER_BLOCK, MAX_VECTOR_OP_BLOCKS))
#define NUM_VECTOR_OP_THREADS_PER_BLOCK(N)  (min((N), MAX_VECTOR_OP_THREADS_PER_BLOCK))

#define PI 3.1415926535897932f

__global__ void kRandomGaussian(unsigned int* rndMults, unsigned long long* rndWords, float* gData, unsigned int numElements) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long rndWord = rndWords[idx];
    const unsigned int rndMult = rndMults[idx];

    float rnd1, rnd2, R, T;
    for(unsigned int i = idx; i < numElements; i += 2*NUM_RND_STREAMS) {
        rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
        rnd1 = ((LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f;
        rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
        rnd2 = ((LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f;
        T = 2 * PI * rnd2;
        R = (-2 * (rnd1));
        gData[i] = R * (T);
        if (i + NUM_RND_STREAMS < numElements)
            gData[i + NUM_RND_STREAMS] = R * __sinf(T);
    }
    rndWords[idx] = rndWord;
}

__global__ void kMinColumnwise(float* mat, float* target, unsigned int width, unsigned int height) {
    __shared__ float min_vals[32];
    float cur_min = 1.0f;
    float val = 0;
 
    for (unsigned int i = threadIdx.x; i < height; i += 32) {
        val = mat[blockIdx.x * height + i];

        if (val < cur_min)
            cur_min = val;
    }

    min_vals[threadIdx.x] = cur_min;

    __syncthreads();

    if (threadIdx.x == 0) {
        cur_min = 1.0f;

        for (unsigned int i = 0; i < 32; i++)
            if (min_vals[i] < cur_min)
                cur_min = min_vals[i];

        target[blockIdx.x] = cur_min;
    }
}
