
#include <hip/hip_runtime.h>
__global__ void g_getCost_3(float* cost,
	float* weight,
	float lambda, int wlen)
{
	__shared__ float _sum[32];
	_sum[threadIdx.x] = 0;
	__syncthreads();
	

	for(int i = 0; i < wlen; i += blockDim.x)
	{
		int id = i + threadIdx.x;
		if(id < wlen)
		{
			_sum[threadIdx.x] += weight[id] * weight[id];
		}
	}

	int len = blockDim.x;
	while(len != 1)
	{
		__syncthreads();
		int skip = (len + 1) >> 1;
		if(threadIdx.x < skip && (threadIdx.x + skip) < len)
		{
			_sum[threadIdx.x] += _sum[threadIdx.x + skip];
		}
		len = skip;
	}
}