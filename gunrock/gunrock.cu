
#include <hip/hip_runtime.h>
#define SizeT int
#define VertexId int

__global__ void Join(
    const SizeT                 edges,
    const SizeT 		iter,
    const SizeT*    const       pos,
    const SizeT*    const	counts,
          SizeT*    	        flag,
    const VertexId* const	intersect,
    const VertexId* const       froms,
    const VertexId* const       tos,
          VertexId*             froms_out,
          VertexId*             tos_out)
{
    SizeT x = blockIdx.x * blockDim.x + threadIdx.x;
  	SizeT tmp = pos[iter];
    SizeT size = ((iter==0) ? tmp:counts[0]) * (pos[iter+1]-tmp);
    if(x>=0 && x<size*edges)
    {
		SizeT a = (x/edges%((iter==0)?tmp:counts[0]))*edges;
		SizeT b = tmp+x/(edges*((iter==0)?tmp:counts[0])); 

		if(iter==0){ 
	    	froms_out[a]=froms[x/edges%tmp];
	    	tos_out[a]=tos[x/edges%tmp];
		}
	    __syncthreads();

		VertexId c = intersect[iter*2];
		VertexId d = intersect[iter*2+1];
		{
			if(c!=0)  
 			{
	    		SizeT edge = c/2; 
	    		if(c%2==1)
	    		{
					if(froms_out[a+edge]!=froms[b]) {
						flag[x/edges]=0; 
						return;
					}
				} else{ 
					if(tos_out[a+edge-1]!=froms[b]) {
						flag[x/edges]=0; 
						return;
					}
				}
			} else{
				for(SizeT edge = 0; edge<iter+1; edge++){
		    		if(froms[b]==froms_out[a+edge] || froms[b]==tos_out[a+edge])
		    		{
		    			flag[x/edges]=0;
		      			return;
		    		}    
				}
			}
		}

		{
			if(d!=0) {
	    		SizeT edge = d/2;
	    		if(d%2==1){
					if(froms_out[a+edge]!=tos[b]) {
						flag[x/edges]=0; 
						return;
					}
				} else{
					if(tos_out[a+edge-1]!=tos[b]) {
						flag[x/edges]=0; 
						return;
					}
				}
			} else {
	    		for(SizeT edge=0; edge<iter+1; edge++) {
	        		if(tos[b]==froms_out[a+edge] || tos[b]==tos_out[a+edge]) {
	    	    		flag[x/edges]=0; 
		    			return;
					}
	    		}
			}
		}
		flag[x/edges]=1;
    }
}