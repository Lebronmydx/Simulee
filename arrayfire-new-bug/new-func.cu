
#include <hip/hip_runtime.h>
#define THREADS_X 16
#define THREADS_Y 16

__global__
void convolve2(int *out_ptr, int *signal_ptr, int nBBS0, int *out_strides, int *out_dims, int *signal_strides, int *signal_dims,
               int nBBS1, int o2, int o3, int s2, int s3, int expand, int fLen0, int fLen1)
{
    const unsigned C_SIZE  = 512;
    __shared__ int shrdMem[512];

    const int radius0  = fLen0-1;
    const int radius1  = fLen1-1;
    const int padding0 = 2*radius0;
    const int padding1 = 2*radius1;
    const int shrdLen0 = THREADS_X + padding0;
    const int shrdLen1 = THREADS_Y + padding1;

    unsigned b0  = blockIdx.x / nBBS0;
    unsigned b1  = (blockIdx.y + blockIdx.z * gridDim.y) / nBBS1;
    int *dst = (int *)out_ptr + (b0 * out_strides[2] + /* activated with batched input signal */
                             o2 * out_strides[2] + /* activated with batched input filter */
                             b1 * out_strides[3] + /* activated with batched input signal */
                             o3 * out_strides[3]); /* activated with batched input filter */

    const int *src = (const int *)signal_ptr + (b0 * signal_strides[2] + /* activated with batched input signal */
                                            s2 * signal_strides[2] + /* activated with batched input filter */
                                            b1 * signal_strides[3] + /* activated with batched input signal */
                                            s3 * signal_strides[3]); /* activated with batched input filter */



    int lx  = threadIdx.x;
    int ly  = threadIdx.y;
    int gx  = THREADS_X * (blockIdx.x-b0*nBBS0) + lx;
    int gy  = THREADS_Y * ((blockIdx.y + blockIdx.z * gridDim.y) -b1*nBBS1) + ly;

    if(b1 >= out_dims[3])
        return;

    int s0 = signal_strides[0];
    int s1 = signal_strides[1];
    int d0 = signal_dims[0];
    int d1 = signal_dims[1];
    // below loops are traditional loops, they only run multiple
    // times filter length is more than launch size
    for (int b=ly, gy2=gy; b<shrdLen1; b+=THREADS_Y, gy2+=THREADS_Y) {
        int j = gy2-radius1;
        bool is_j  = j>=0 && j<d1;
        // move row_set THREADS_Y along coloumns
        for (int a=lx, gx2=gx; a<shrdLen0; a+=THREADS_X, gx2+=THREADS_X) {
            int i = gx2-radius0;
            bool is_i  = i>=0 && i<d0;
            shrdMem[b*shrdLen0+a] = (is_i && is_j ? src[i*s0+j*s1] : 0);
        }
    }
    __syncthreads();

    if (gx<out_dims[0] && gy<out_dims[1]) {
        int ci = lx + radius0 + (expand ? 0 : fLen0>>1);
        int cj = ly + radius1 + (expand ? 0 : fLen1>>1);

        int accum = 0;
        for(int fj=0; fj<fLen1; ++fj) {
            for(int fi=0; fi<fLen0; ++fi) {

                int s_val = shrdMem[(cj-fj)*shrdLen0 + (ci-fi)];
                accum   = accum + s_val;
            }
        }
        dst[gy*out_strides[1]+gx] = (int)accum;
    }
}