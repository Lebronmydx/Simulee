
#include <hip/hip_runtime.h>
#define CU1DBLOCK 256

__global__
void _copy_low_upp(float* A, int rows, int stride) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i <= j || i >= rows)
    return;
  int index_1 = i * stride + j;
  int index_2 = j * stride + i;
  A[index_2] = A[index_1];
}
// rows = 5, stride = 0, block = (2, 1, 1), thread = (3, 2, 2)
// (0, 0, 0) (1, 0, 1) with (0, 0, 0) (1, 0, 0)
// i = 1, j = 0, index_1 = 0, index_2 = 1; i = 1, j = 0, index_1 = 0, index_2 = 1
// (2, 1, 0) read, (1, 0, 1) write. (1 0 1) write to 1, (2 1 0) read from 1

// i = 2, j = 1, index_1 = 7, index_2 = 5
// i = 5, j = 0, index_1 = 15, index_2 = 5



__global__
void _copy_upp_low(float* A, int rows, int stride) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (j <= i || j >= rows)
    return;
  int index_1 = i * stride + j;
  int index_2 = j * stride + i;
  A[index_2] = A[index_1];
}


__global__
void _add_diag_vec_mat(float alpha, float *mat, int stride, int rows, int cols,
                              const float *vec, const float *mat2,
                              int mat2_row_stride, int mat2_col_stride,
                              float beta) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;  
  int j = blockIdx.y * blockDim.y + threadIdx.y;  

  int index = j * stride + i, index2 = j * mat2_row_stride
      + i * mat2_col_stride;

  if (i < cols && j < rows) {
    mat[index] = alpha * vec[j] * mat2[index2] + beta * mat[index];
  }
}

__global__
void _copy_from_tp(float* A, const float* B, int dmat_cols, int dmat_rows, int dmat_stride) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;  // col index
  int j = blockIdx.y * blockDim.y + threadIdx.y;  // row index
  if (i < dmat_cols && j < dmat_rows) {
    int index_B = (j * (j + 1) / 2) + i;
    int index_A = j * dmat_stride + i;
    if (i <= j) {
      A[index_A] = B[index_B];
    } else {
      A[index_A] = 0.0;
    }
  }
}


__global__
void _copy_from_mat(float* mat_out, const float* mat_in,
                           int d_out_stride, int d_out_rows, int d_out_cols, int d_in_stride) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;  // col-index
  int j = blockIdx.y * blockDim.y + threadIdx.y;  // row-index.
  int index_out = i + j * d_out_stride;
  int index_in = i + j * d_in_stride;
  if (i < d_out_cols && j < d_out_rows)
    mat_out[index_out] = mat_in[index_in];
}


__global__
void _trace_mat_mat_trans(const float* A, const float* B, int dA_rows, int dA_cols, int dA_stride,
                                 int B_stride, float* value) {
  __shared__ float ssum[CU1DBLOCK];
  // linear thread id;
  const int tid = threadIdx.y * blockDim.x + threadIdx.x;
  const int j = blockIdx.x * blockDim.x + threadIdx.x;
  const int grid_height = gridDim.y * blockDim.y;
  int i = blockIdx.y * blockDim.y + threadIdx.y;

  // Grid reduce
  float tsum = 0.0;
  if (j < dA_cols) {
    while (i < dA_rows) {
      tsum += A[i * dA_stride + j] * B[i * B_stride + j];
      i += grid_height;
    }
  }
  ssum[tid] = tsum;
  __syncthreads();

  // Block reduce
  for (int shift = CU1DBLOCK / 2; shift > warpSize; shift >>= 1) {
    if (tid < shift)
      ssum[tid] += ssum[tid + shift];
    __syncthreads();
  }

  // Warp reduce. Implicitly synchronized within a warp.
  if (tid < warpSize) {
    for (int shift = warpSize; shift > 0; shift >>= 1) {
      ssum[tid] += ssum[tid + shift];
    }
  }

  // output 1 sum per thread block
  if (tid == 0) {
    value[blockIdx.y * gridDim.x + blockIdx.x] = ssum[0];
  }
}


__global__
void _splice(float* y, const float* x, const int* off,
                    int d_out_cols, int d_out_rows, int d_out_stride, 
					int d_in_cols, int d_in_rows, int d_in_stride) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int index = i + j * d_out_stride;
  if (i < d_out_cols && j < d_out_rows) {
    int src_col = i % d_in_cols;
    int src_row = j + off[i / d_in_cols];
    if (src_row < 0)
      src_row = 0;
    if (src_row >= d_in_rows)
      src_row = d_in_rows - 1;
    y[index] = x[src_col + src_row * d_in_stride];
  }
}