
#include <hip/hip_runtime.h>
__global__ void MatchSiftPoints(float *sift1, float *sift2, float *corrData, int numPts1, int numPts2)
{
  __shared__ float siftPoint[128];
  __shared__ float sums[16*16];
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int p1 = blockIdx.x;
  const int p2 = blockIdx.y*16 + ty;
  const float *ptr1 = sift1;
  const float *ptr2 = sift2;
  const int i = 16*ty + tx;
  if (ty<8)
    siftPoint[i] = ptr1[i];
  __syncthreads();
  float sum = 0.0f;
  if (p2<numPts2)
    for (int j=0;j<8;j++)
      sum += siftPoint[16*j+tx] * ptr2[16*j+tx];
  sums[i] = sum;
  __syncthreads();
  if (tx<8)
    sums[i] += sums[i+8];
  __syncthreads();
  if (tx<4)
    sums[i] += sums[i+4];
  __syncthreads();
  if (ty==0) {
    sum = sums[16*tx+0] + sums[16*tx+1] + sums[16*tx+2] + sums[16*tx+3];
    corrData[p1*gridDim.y*16 + blockIdx.y*16 + tx] = sum;
  }
  __syncthreads();
}

__global__ void MatchSiftPoints2(float *sift1, float *sift2, float *corrData, int numPts1, int numPts2)
{
  __shared__ float siftPoints1[16*128];
  __shared__ float siftPoints2[16*128];
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const float *ptr1 = sift1;
  const float *ptr2 = sift2;
  for (int i=0;i<8;i++) {
    siftPoints1[128*ty+16*i+tx] = ptr1[16*i+tx];
    siftPoints2[128*ty+16*i+tx] = ptr2[16*i+tx];
  }
  __syncthreads();
  const int p1 = blockIdx.x*16 + ty;
  const int p2 = blockIdx.y*16 + tx;
  const float *pt1 = &siftPoints1[ty*128];
  const float *pt2 = &siftPoints2[tx*128];
  float sum = 0.0f;
  for (int i=0;i<128;i++) {
    int itx = (i + tx)&127; // avoid bank conflicts
    sum += pt1[itx]*pt2[itx];
  }
  if (p1<numPts1)
    corrData[p1*gridDim.y*16 + p2] = (p2<numPts2 ? sum : -1.0f);
}


__global__ void MatchSiftPoints3(float *sift1, float *sift2, float *corrData, int numPts1, int numPts2)
{
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int p1 = blockIdx.x*16 + ty;
  const int p2 = blockIdx.y*16 + tx;
  const float *pt1 = sift1;
  const float *pt2 = sift2;
  float sum = 0.0f;
  for (int i=0;i<128;i++) 
    sum += pt1[i]*pt2[i];
  if (p1<numPts1)
    corrData[p1*gridDim.y*16 + p2] = (p2<numPts2 ? sum : -1.0f);
}

__global__ void MatchSiftPoints4(float *sift1, float *sift2, float *corrData, int numPts1, int numPts2)
{
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int p1 = blockIdx.x;
  const int p2 = blockIdx.y*16 + ty;
  const float *ptr1 = sift1;
  const float *ptr2 = sift2;
  float sum = 0.0f;
  if (p2<numPts2)
    for (int j=0;j<8;j++)
      sum += ptr1[16*j+tx] * ptr2[16*j+tx];
  if (tx==0)
    corrData[p1*gridDim.y*16 + blockIdx.y*16 + ty] = sum;
}

__global__ void FindMaxCorr(float *corrData, float *sift1, float *sift2, int numPts1, int corrWidth, int siftSize)
{
  __shared__ float maxScore[16*16];
  __shared__ float maxScor2[16*16];
  __shared__ int maxIndex[16*16];
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int idx = ty*16 + tx;
  int p1 = blockIdx.x*16 + threadIdx.y;
  p1 = (p1>=numPts1 ? numPts1-1 : p1);
  maxScore[idx] = -1.0f;
  maxScor2[idx] = -1.0f;
  maxIndex[idx] = -1;
  __syncthreads();
  float *corrs = &corrData[p1*corrWidth];
  for (int i=tx;i<corrWidth;i+=16) {
    float val = corrs[i];
    if (val>maxScore[idx]) {
      maxScor2[idx] = maxScore[idx];
      maxScore[idx] = val;
      maxIndex[idx] = i;
    } else if (val>maxScor2[idx])
      maxScor2[idx] = val;
  }
  __syncthreads();
  for (int len=8;len>0;len/=2) {
    if (tx<8) {
      float val = maxScore[idx+len];
      int i = maxIndex[idx+len];
      if (val>maxScore[idx]) {
	maxScor2[idx] = maxScore[idx];
	maxScore[idx] = val;
	maxIndex[idx] = i;
      } else if (val>maxScor2[idx])
	maxScor2[idx] = val;
      float va2 = maxScor2[idx+len];
      if (va2>maxScor2[idx])
	maxScor2[idx] = va2;
    }
    __syncthreads();
  }
  if (tx==6)
    sift1[p1] = maxScore[ty*16];
  if (tx==7)
    sift1[p1] = maxScor2[ty*16] / (maxScore[ty*16] + 1e-6);
  if (tx==8)
    sift1[p1] = maxIndex[ty*16];
  if (tx==9)
    sift1[p1] = sift2[maxIndex[ty*16]];
  if (tx==10)
    sift1[p1] = sift2[maxIndex[ty*16]];
  __syncthreads();
}


__global__ void FindMaxCorr_2(float *corrData, float *sift1, float *sift2, int numPts1, int corrWidth, int siftSize)
{
  __shared__ float maxScore[16*16];
  __shared__ float maxScor2[16*16];
  __shared__ int maxIndex[16*16];
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int idx = ty*16 + tx;
  int p1 = blockIdx.x*16 + threadIdx.y;
  p1 = (p1>=numPts1 ? numPts1-1 : p1);
  maxScore[idx] = -1.0f;
  maxScor2[idx] = -1.0f;
  maxIndex[idx] = -1;
  __syncthreads();
  float *corrs = &corrData[p1*corrWidth];
  for (int i=tx;i<corrWidth;i+=16) {
    float val = corrs[i];
    if (val>maxScore[idx]) {
      maxScor2[idx] = maxScore[idx];
      maxScore[idx] = val;
      maxIndex[idx] = i;
    } else if (val>maxScor2[idx])
      maxScor2[idx] = val;
  }
  __syncthreads();
  for (int len=8;len>0;len/=2) {
    if (tx<8) {
      float val = maxScore[idx+len];
      int i = maxIndex[idx+len];
      if (val>maxScore[idx]) {
	maxScor2[idx] = maxScore[idx];
	maxScore[idx] = val;
	maxIndex[idx] = i;
      } else if (val>maxScor2[idx])
	maxScor2[idx] = val;
      float va2 = maxScor2[idx+len];
      if (va2>maxScor2[idx])
	maxScor2[idx] = va2;
    }
    __syncthreads();
  }
  if (tx==6)
    sift1[p1] = maxScore[ty*16];
  if (tx==7)
    sift1[p1] = maxScor2[ty*16] / (maxScore[ty*16] + 1e-6);
  if (tx==8)
    sift1[p1] = maxIndex[ty*16];
  if (tx==9)
    sift1[p1] = sift2[maxIndex[ty*16]];
  if (tx==10)
    sift1[p1] = sift2[maxIndex[ty*16]];
}