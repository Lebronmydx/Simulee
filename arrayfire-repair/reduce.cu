
#include <hip/hip_runtime.h>
/* arrayfire: Fix race condition in reduce_first_kernel.*/
    __global__ void warp_reduce(double *s_ptr)
    {
        int tidx = threadIdx.x;
        int tidy = threadIdx.y;
        double *s_ptr_vol = s_ptr + tidx;
        double tmp = *s_ptr;

        for (int n = 16; n >= 1; n >>= 1) {
            if (tidx < n) {
                double val1, val2;
                val1 = s_ptr_vol[0];
                val2 = s_ptr_vol[n];

                tmp = val1 + val2;
                s_ptr_vol[0] = tmp;
            }
        }
    }

