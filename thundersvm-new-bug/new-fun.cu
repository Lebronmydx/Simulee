
#include <hip/hip_runtime.h>
    __host__ __device__ inline bool is_I_up(float a, float y, float Cp, float Cn) {
        return (y > 0 && a < Cp) || (y < 0 && a > 0);
    }

    __host__ __device__ inline bool is_I_low(float a, float y, float Cp, float Cn) {
        return (y > 0 && a > 0) || (y < 0 && a < Cn);
    }

    __host__ __device__ inline bool is_free(float a, float y, float Cp, float Cn) {
        return a > 0 && (y > 0 ? a < Cp : a < Cn);
    }
	
	__host__ __device__ inline bool min_t(float a, float y) {
        return a > y ? y : a;
    }

	__host__ __device__ inline bool max_t(float a, float y) {
        return a > y ? a : y;
    }
	
    __device__ int get_block_min_t(const float *values, int *index) {
        int tid = threadIdx.x;
        index[tid] = tid;
        __syncthreads();
        //block size is always the power of 2
        for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
            if (tid < offset) {
                if (values[index[tid + offset]] < values[index[tid]]) {
                    index[tid] = index[tid + offset];
                }
            }
            __syncthreads();
        }
        return index[0];
    }


    __global__ void
    c_smo_solve_kernel(const int *label, float *f_val, float *alpha, float *alpha_diff,
                       const int *working_set, int ws_size,
                       float Cp, float Cn, const float *k_mat_rows, const float *k_mat_diag, int row_len,
                       float eps,
                       float *diff, int max_t_iter) {
        //"row_len" equals to the number of instances in the original training dataset.
        //allocate shared memory
        __shared__ int shared_mem[256];
        int *f_idx2reduce = shared_mem; //temporary memory for reduction
        float *f_val2reduce = (float *) &shared_mem[ws_size]; //f values used for reduction.
        float *alpha_i_diff = (float *) &shared_mem[ws_size + ws_size * sizeof(float) / sizeof(int)]; //delta alpha_i
        float *alpha_j_diff = &alpha_i_diff[1];
        float *kd = (float *) &alpha_j_diff[1]; // diagonal elements for kernel matrix

        //index, f value and alpha for each instance
        int tid = threadIdx.x;
        int wsi = working_set[tid];
        kd[tid] = k_mat_diag[wsi];
        float y = label[wsi];
        float f = f_val[wsi];
        float a = alpha[wsi];
        float aold = a;
        __syncthreads();
        float local_eps;
        int numOfIter = 0;
        while (1) {
            //select fUp and fLow
            if (is_I_up(a, y, Cp, Cn))
                f_val2reduce[tid] = f;
            else
                f_val2reduce[tid] = INFINITY;
            int i = get_block_min_t(f_val2reduce, f_idx2reduce);
            float up_value = f_val2reduce[i];
            float kIwsI = k_mat_rows[row_len * i + wsi];//K[i, wsi]
            __syncthreads();

            if (is_I_low(a, y, Cp, Cn))
                f_val2reduce[tid] = -f;
            else
                f_val2reduce[tid] = INFINITY;
            int j1 = get_block_min_t(f_val2reduce, f_idx2reduce);
            float low_value = -f_val2reduce[j1];

            float local_diff = low_value - up_value;
            if (numOfIter == 0) {
                local_eps = max_t(eps, 0.1f * local_diff);
                if (tid == 0) {
                    diff[0] = local_diff;
                }
            }

            if (numOfIter > max_t_iter || local_diff < local_eps) {
                alpha[wsi] = a;
                alpha_diff[tid] = -(a - aold) * y;
                diff[1] = numOfIter;
                break;
            }
            __syncthreads();

            //select j2 using second order heuristic
            if (-up_value > -f && (is_I_low(a, y, Cp, Cn))) {
                float aIJ = kd[i] + kd[tid] - 2 * kIwsI;
                float bIJ = -up_value + f;
                f_val2reduce[tid] = (-bIJ * bIJ / aIJ);
            } else
                f_val2reduce[tid] = INFINITY;
            int j2 = get_block_min_t(f_val2reduce, f_idx2reduce);

            //update alpha
            if (tid == i)
                *alpha_i_diff = y > 0 ? Cp - a : a;
            if (tid == j2)
                *alpha_j_diff = min_t(y > 0 ? a : Cn - a, (-up_value + f) / (kd[i] + kd[j2] - 2 * kIwsI));
            __syncthreads();
            float l = min_t(*alpha_i_diff, *alpha_j_diff);

            if (tid == i)
                a += l * y;
            if (tid == j2)
                a -= l * y;

            //update f
            float kJ2wsI = k_mat_rows[row_len * j2 + wsi];//K[J2, wsi]
            f -= l * (kJ2wsI - kIwsI);
            numOfIter++;
        }
    }