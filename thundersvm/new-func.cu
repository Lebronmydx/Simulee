#include "hip/hip_runtime.h"
#define KERNEL_LOOP(i, n) \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
       i < (n); \
       i += blockDim.x * gridDim.x)


    __global__ void
    update_f_kernel(float_type *f, int ws_size, const float_type *alpha_diff, const kernel_type *k_mat_rows,
                    int n_instances) {
        //"n_instances" equals to the number of rows of the whole kernel matrix for both SVC and SVR.
        KERNEL_LOOP(idx, n_instances) {//one thread to update multiple fvalues.
            double sum_diff = 0;
            for (int i = 0; i < ws_size; ++i) {
                double d = alpha_diff[i];
                if (d != 0) {
                    sum_diff += d * k_mat_rows[i * n_instances + idx];
                }
            }
            f[idx] -= sum_diff;
        }
    }

